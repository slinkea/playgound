#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "CudaKernels.cuh"


__global__ void Kernel(int n, short* a)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < n) {
    a[tid]++;
  }
}

CudaKernel::CudaKernel()
{
  std::cout << "Reading CUDA-capable GPUs properties... " << std::endl;

  int deviceQty{};
  bool success{};
  if (hipGetDeviceCount(&deviceQty) == hipError_t::hipSuccess) {
    for (int deviceIdx{}; deviceIdx < deviceQty; deviceIdx++) {
      hipDeviceProp_t prop;
      if (hipGetDeviceProperties(&prop, deviceIdx) == hipError_t::hipSuccess) {
        std::cout << "Device Number: " << deviceIdx << std::endl;
        std::cout << "  Device name: " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "  Peak Memory Bandwidth (GB/s): " << (2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6) << std::endl;
        std::cout << "  Global Memory (Bytes): " << prop.totalGlobalMem << std::endl;
        std::cout << std::endl;

        success = true;
      }
    }
  }

  if (!success) {
    throw std::runtime_error("Error while using CUDA library.");
  }



  //int N = 20 * (1 << 20);
  //float* x, * y, * d_x, * d_y;
  //x = (float*)malloc(N * sizeof(float));
  //y = (float*)malloc(N * sizeof(float));

  //hipMalloc(&d_x, N * sizeof(float));
  //hipMalloc(&d_y, N * sizeof(float));

  //for (int i = 0; i < N; i++) {
  //  x[i] = 1.0f;
  //  y[i] = 2.0f;
  //}

  //hipEvent_t start, stop;
  //hipEventCreate(&start);
  //hipEventCreate(&stop);

  //hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  //hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  //hipEventRecord(start);

  //// Perform Kernel on 1M elements
  //Kernel<<<(N + 511) / 512, 512>>>(N, 2.0f, d_x, d_y);

  //hipEventRecord(stop);

  //hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  //hipEventSynchronize(stop);
  //float milliseconds = 0;
  //hipEventElapsedTime(&milliseconds, start, stop);

  //float maxError = 0.0f;
  //for (int i = 0; i < N; i++) {
  //  maxError = std::max(maxError, abs(y[i] - 4.0f));
  //}

  //std::cout << "Max error: " << maxError << std::endl;
  //std::cout << "Effective Bandwidth (GB/s): " << (N * 4 * 3 / milliseconds / 1e6) << std::endl;
}

void CudaKernel::MemoryAllocation()
{
  // Allocate pinned memory
  bool success{};
  if (hipHostMalloc(&m_memHost, 1 << 30) == hipError_t::hipSuccess) {
    if (hipMalloc(&m_memDevice, 1 << 30) == hipError_t::hipSuccess) {
      success = true;
    }
  }

  if (!success) {
    throw std::runtime_error("Error while allocating memory.");
  }
}

void CudaKernel::CopyMemoryToDevice()
{
  hipError_t error = hipMemcpy(m_memDevice, m_memHost, 1 << 30, hipMemcpyHostToDevice);
}

void CudaKernel::Process()
{
  int N = 1 << 30;
  int blockSize = 0;
  int gridSize = (N + 512 - 1) / 512;
  dim3 blocks(gridSize, 1, 1);
  dim3 threadPerBlock(blockSize, 1, 1);

  Kernel<<<blocks, threadPerBlock>>>(N, m_memDevice);
  if (hipDeviceSynchronize() == hipError_t::hipSuccess) {
    std::cout << "done" << std::endl;

    //hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
  }
}

CudaKernel::~CudaKernel()
{
  bool success{};
  if (hipHostFree(m_memHost) == hipError_t::hipSuccess) {
    if (hipFree(m_memDevice) == hipError_t::hipSuccess) {
      success = true;
    }
  }

  if (!success) {
    std::cout << "Error while freeing memory." << std::endl;
  }
}
